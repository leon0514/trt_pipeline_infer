#include "hip/hip_runtime.h"
#include "kernels/kernel_warp.hpp"
#include "trt/sahiyolo/yolo11_obb_sahi.hpp"
namespace sahiyolo
{

bool Yolo11ObbSahiModelImpl::load(const std::string &engine_file,
                               const std::vector<std::string> &names,
                               float confidence_threshold,
                               float nms_threshold,
                               int gpu_id,
                               int max_batch_size,
                               bool auto_slice,
                               int slice_width,
                               int slice_height,
                               double slice_horizontal_ratio,
                               double slice_vertical_ratio)
{
    trt_       = TensorRT::load(engine_file);
    device_id_ = gpu_id;
    if (trt_ == nullptr)
        return false;

    trt_->print();

    this->slice_ = std::make_shared<slice::SliceImage>();

    this->num_box_element_        = 10;
    this->confidence_threshold_   = confidence_threshold;
    this->nms_threshold_          = nms_threshold;
    this->class_names_            = names;
    this->max_batch_size_         = max_batch_size;
    this->auto_slice_             = auto_slice;
    this->slice_width_            = slice_width;
    this->slice_height_           = slice_height;
    this->slice_horizontal_ratio_ = slice_horizontal_ratio;
    this->slice_vertical_ratio_   = slice_vertical_ratio;

    auto input_dim  = trt_->static_dims(0);
    bbox_head_dims_ = trt_->static_dims(1);

    network_input_width_  = input_dim[3];
    network_input_height_ = input_dim[2];
    isdynamic_model_      = trt_->has_dynamic_dim();

    normalize_   = norm_image::Norm::alpha_beta(1 / 255.0f, 0.0f, norm_image::ChannelType::SwapRB);
    num_classes_ = bbox_head_dims_[2] - 5;
    return true;
}



InferResult Yolo11ObbSahiModelImpl::forwards(const std::vector<cv::Mat> &inputs, void *stream)
{
    checkRuntime(hipSetDevice(device_id_));
    assert(inputs.size() == 1);

    if (auto_slice_)
    {
        slice_->autoSlice(tensor::Image(inputs[0].data, inputs[0].cols, inputs[0].rows));
    }
    else
    {
        slice_->slice(tensor::Image(inputs[0].data, inputs[0].cols, inputs[0].rows),
                      slice_width_,
                      slice_height_,
                      slice_horizontal_ratio_,
                      slice_vertical_ratio_,
                      stream);
    }

    int num_image          = slice_->slice_num_h_ * slice_->slice_num_v_;
    this->max_image_boxes_ = single_image_max_boxes_ * num_image;
    auto input_dims        = trt_->static_dims(0);
    int infer_batch_size   = input_dims[0];
    if (infer_batch_size != num_image)
    {
        if (isdynamic_model_)
        {
            assert(num_image <= max_batch_size_);
            infer_batch_size = num_image;
            input_dims[0]    = num_image;
            if (!trt_->set_run_dims(0, input_dims))
            {
                printf("Fail to set run dims\n");
                return {};
            }
        }
        else
        {
            if (infer_batch_size < num_image)
            {
                printf("When using static shape model, number of images[%d] must be "
                       "less than or equal to the maximum batch[%d].",
                       num_image,
                       infer_batch_size);
                return {};
            }
        }
    }
    adjust_memory(infer_batch_size);

    // 每一张小图的尺寸都是一致的，所以只需要取计算一次仿射矩阵
    affine::LetterBoxMatrix affine_matrix;
    hipStream_t stream_ = (hipStream_t)stream;
    compute_affine_matrix(affine_matrix, stream_);
    for (int i = 0; i < num_image; ++i)
    {
        preprocess(i, stream);
    }

    float *bbox_output_device = bbox_predict_.gpu();
#if NV_TENSORRT_MAJOR >= 10
    // yolov5 模型推理
    // TensorRT10需要指定输入输出名字，这里的输入输出分别是images, output0
    std::unordered_map<std::string, const void *> bindings = {{"images", input_buffer_.gpu()},
                                                              {"output0", bbox_output_device}};
    if (!trt_->forward(bindings, stream_))
    {
        printf("Failed to tensorRT forward.\n");
        return {};
    }
#else
    std::vector<void *> bindings{input_buffer_.gpu(), bbox_output_device};
    if (!trt_->forward(bindings, stream_))
    {
        printf("Failed to tensorRT forward.\n");
        return {};
    }
#endif

    int *box_count = image_box_count_.gpu();
    checkRuntime(hipMemsetAsync(box_count, 0, sizeof(int), stream_));

    for (int ib = 0; ib < num_image; ++ib)
    {
        int start_x                    = slice_->slice_start_point_.cpu()[ib * 2];
        int start_y                    = slice_->slice_start_point_.cpu()[ib * 2 + 1];
        float *boxarray_device         = output_boxarray_.gpu();
        float *affine_matrix_device    = affine_matrix_.gpu();
        float *image_based_bbox_output = bbox_output_device + ib * (bbox_head_dims_[1] * bbox_head_dims_[2]);
        decode_kernel_invoker_v11_obb(image_based_bbox_output,
                                  bbox_head_dims_[1],
                                  num_classes_,
                                  bbox_head_dims_[2],
                                  confidence_threshold_,
                                  nms_threshold_,
                                  affine_matrix_device,
                                  boxarray_device,
                                  box_count,
                                  max_image_boxes_,
                                  num_box_element_,
                                  start_x,
                                  start_y,
                                  ib,
                                  stream_);
    }

    float *boxarray_device = output_boxarray_.gpu();
    fast_nms_kernel_invoker_v11_obb(boxarray_device, box_count, max_image_boxes_, nms_threshold_, num_box_element_, stream_);

    checkRuntime(hipMemcpyAsync(output_boxarray_.cpu(),
                                 output_boxarray_.gpu(),
                                 output_boxarray_.gpu_bytes(),
                                 hipMemcpyDeviceToHost,
                                 stream_));
    checkRuntime(hipMemcpyAsync(image_box_count_.cpu(),
                                 image_box_count_.gpu(),
                                 image_box_count_.gpu_bytes(),
                                 hipMemcpyDeviceToHost,
                                 stream_));
    checkRuntime(hipStreamSynchronize(stream_));

    std::vector<object::DetectionObbResultArray> arrout(1);
    for (int ib = 0; ib < 1; ++ib)
    {
        float *parray                        = output_boxarray_.cpu();
        int count                            = min(max_image_boxes_, *(image_box_count_.cpu()));
        object::DetectionObbResultArray &output = arrout[ib];
        for (int i = 0; i < count; ++i)
        {
            float *pbox      = parray + i * num_box_element_;
            int label        = pbox[6];
            int keepflag     = pbox[7];
            std::string name = class_names_[label];
            if (keepflag == 1)
            {
                object::OBBox result_object_box(pbox[0], pbox[1], pbox[2], pbox[3], pbox[4], pbox[5], label, name);
                output.emplace_back(std::move(result_object_box));
            }
        }
    }
    return arrout;
}

static InferBase *loadraw(const std::string &engine_file,
                          const std::vector<std::string> &names,
                          float confidence_threshold,
                          float nms_threshold,
                          int gpu_id,
                          int max_batch_size,
                          bool auto_slice,
                          int slice_width,
                          int slice_height,
                          double slice_horizontal_ratio,
                          double slice_vertical_ratio)
{
    Yolo11ObbSahiModelImpl *impl = new Yolo11ObbSahiModelImpl();
    if (!impl->load(engine_file,
                    names,
                    confidence_threshold,
                    nms_threshold,
                    gpu_id,
                    max_batch_size,
                    auto_slice,
                    slice_width,
                    slice_height,
                    slice_horizontal_ratio,
                    slice_vertical_ratio))
    {
        delete impl;
        impl = nullptr;
    }
    return impl;
}

std::shared_ptr<InferBase> load_yolo_11_obb_sahi(const std::string &engine_file,
                                             const std::vector<std::string> &names,
                                             int gpu_id,
                                             float confidence_threshold,
                                             float nms_threshold,
                                             int max_batch_size,
                                             bool auto_slice,
                                             int slice_width,
                                             int slice_height,
                                             double slice_horizontal_ratio,
                                             double slice_vertical_ratio)
{
    try
    {
        checkRuntime(hipSetDevice(gpu_id));
        return std::shared_ptr<Yolo11ObbSahiModelImpl>((Yolo11ObbSahiModelImpl *)loadraw(engine_file,
                                                                                   names,
                                                                                   confidence_threshold,
                                                                                   nms_threshold,
                                                                                   gpu_id,
                                                                                   max_batch_size,
                                                                                   auto_slice,
                                                                                   slice_width,
                                                                                   slice_height,
                                                                                   slice_horizontal_ratio,
                                                                                   slice_vertical_ratio),
                                                    [](InferBase *impl) { delete impl; });
    }
    catch (const std::exception &ex)
    {
        return nullptr;
    }
}

} // namespace sahiyolo